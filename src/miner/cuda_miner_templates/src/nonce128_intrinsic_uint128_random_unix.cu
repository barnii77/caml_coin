// NVRTC-OPTIONS: --device-int128
/*
 * Copied from:
 * https://github.com/mochimodev/cuda-hashing-algos/blob/master/sha256.cu
 *
 * sha256.cu Implementation of SHA256 Hashing
 *
 * Date: 12 June 2019
 * Revision: 1
 * *
 * Based on the public domain Reference Implementation in C, by
 * Brad Conte, original code here:
 *
 * https://github.com/B-Con/crypto-algorithms
 *
 * This file is released into the Public Domain.
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

typedef unsigned char BYTE;
typedef unsigned int WORD;
typedef unsigned long long LONG;
typedef unsigned __int128 uint128_t;

#define SHA256_BLOCK_SIZE 32  // SHA256 outputs a 32 byte digest

#ifndef MAX_VALID_HASH_LOCATION
#define MAX_VALID_HASH_LOCATION \
  2  // 0 = global mem, 1 = shared mem, >=2 = registers
#endif

typedef struct {
    BYTE data[64];
    WORD datalen;
    LONG bitlen;
    WORD state[8];
} CUDA_SHA256_CTX;

typedef struct __align__(8 * sizeof(WORD)) uint256_t {
    WORD _0;
    WORD _1;
    WORD _2;
    WORD _3;
    WORD _4;
    WORD _5;
    WORD _6;
    WORD _7;
} uint256_t;

#ifndef ROTLEFT
#define ROTLEFT(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
#endif

#define ROTRIGHT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))

#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))

__constant__ WORD k[64] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1,
        0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
        0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, 0xe49b69c1, 0xefbe4786,
        0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147,
        0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
        0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b,
        0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a,
        0x5b9cca4f, 0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
        0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__device__ __forceinline__ void cuda_sha256_transform(CUDA_SHA256_CTX *ctx,
                                                      const BYTE data[]) {
    WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

#pragma unroll 16
    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) |
               (data[j + 3]);
#pragma unroll 48
    for (; i < 64; ++i)
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

    a = ctx->state[0];
    b = ctx->state[1];
    c = ctx->state[2];
    d = ctx->state[3];
    e = ctx->state[4];
    f = ctx->state[5];
    g = ctx->state[6];
    h = ctx->state[7];

#pragma unroll 64
    for (i = 0; i < 64; ++i) {
        t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    ctx->state[0] += a;
    ctx->state[1] += b;
    ctx->state[2] += c;
    ctx->state[3] += d;
    ctx->state[4] += e;
    ctx->state[5] += f;
    ctx->state[6] += g;
    ctx->state[7] += h;
}

__device__ void cuda_sha256_init(CUDA_SHA256_CTX *ctx) {
    ctx->datalen = 0;
    ctx->bitlen = 0;
    ctx->state[0] = 0x6a09e667;
    ctx->state[1] = 0xbb67ae85;
    ctx->state[2] = 0x3c6ef372;
    ctx->state[3] = 0xa54ff53a;
    ctx->state[4] = 0x510e527f;
    ctx->state[5] = 0x9b05688c;
    ctx->state[6] = 0x1f83d9ab;
    ctx->state[7] = 0x5be0cd19;
}

__device__ void cuda_sha256_update(CUDA_SHA256_CTX *ctx, const BYTE data[],
                                   size_t len) {
    WORD i;

    for (i = 0; i < len; ++i) {
        ctx->data[ctx->datalen] = data[i];
        ctx->datalen++;
        if (ctx->datalen == 64) {
            cuda_sha256_transform(ctx, ctx->data);
            ctx->bitlen += 512;
            ctx->datalen = 0;
        }
    }
}

__device__ void cuda_sha256_final(CUDA_SHA256_CTX *ctx, BYTE hash[]) {
    WORD i;

    i = ctx->datalen;

    // Pad whatever data is left in the buffer.
    if (ctx->datalen < 56) {
        ctx->data[i++] = 0x80;
        while (i < 56) ctx->data[i++] = 0x00;
    } else {
        ctx->data[i++] = 0x80;
        while (i < 64) ctx->data[i++] = 0x00;
        cuda_sha256_transform(ctx, ctx->data);
        memset(ctx->data, 0, 56);
    }

    // Append to the padding the total message's length in bits and transform.
    ctx->bitlen += ctx->datalen * 8;
    ctx->data[63] = ctx->bitlen;
    ctx->data[62] = ctx->bitlen >> 8;
    ctx->data[61] = ctx->bitlen >> 16;
    ctx->data[60] = ctx->bitlen >> 24;
    ctx->data[59] = ctx->bitlen >> 32;
    ctx->data[58] = ctx->bitlen >> 40;
    ctx->data[57] = ctx->bitlen >> 48;
    ctx->data[56] = ctx->bitlen >> 56;
    cuda_sha256_transform(ctx, ctx->data);

    // normally sha256 outputs big endian, but here we output little endian
#pragma unroll 4
    for (i = 0; i < 4; ++i) {
        hash[i] = ctx->state[7] >> i * 8;
        hash[i + 4] = ctx->state[6] >> i * 8;
        hash[i + 8] = ctx->state[5] >> i * 8;
        hash[i + 12] = ctx->state[4] >> i * 8;
        hash[i + 16] = ctx->state[3] >> i * 8;
        hash[i + 20] = ctx->state[2] >> i * 8;
        hash[i + 24] = ctx->state[1] >> i * 8;
        hash[i + 28] = ctx->state[0] >> i * 8;
    }
}

/// checks if a 256 bit little endian integer is less than another
__forceinline__ __device__ bool cuda_u256_lte(uint256_t const a, uint256_t const b) {
#pragma unroll sizeof(uint256_t) / sizeof(WORD)
    for (int i = sizeof(uint256_t) / sizeof(WORD) - 1; i >= 0; i--) {
        WORD a_ = ((WORD *) &a)[i], b_ = ((WORD *) &b)[i];
        // NOTE: first checking a_ > b_ is more efficient because most nonces are
        // invalid and therefore if a_ > b_ is the most common exit path (very often
        // in the first loop iteration), thus avoiding almost all computations
        if (a_ > b_) return false;
        if (a_<b_) return true;
    }
    return true;
}

/// mines until any(nonce_out[i] != 0 for i in range(0, sizeof(uint128_t)))
extern "C" __global__ void mine_sha256(BYTE const const_in[SHA256_BLOCK_SIZE],
                                       BYTE nonce_out[sizeof(uint128_t)],
                                       WORD const nonce_step_size,
                                       WORD const n_batch_device,
                                       BYTE const max_valid_hash[SHA256_BLOCK_SIZE],
                                       BYTE const init_nonce[sizeof(uint128_t)]) {
    hiprandStateXORWOW_t state;
    WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(clock64(), thread, 0, &state);

#if MAX_VALID_HASH_LOCATION == 0  // leave it in global mem
    uint256_t _max_valid_hash = *((uint256_t *)max_valid_hash);
#elif MAX_VALID_HASH_LOCATION == 1  // put it into shared mem
    __shared__ uint256_t _max_valid_hash;
    if (threadIdx.x < SHA256_BLOCK_SIZE / sizeof(WORD)) {
        // populate _max_valid_hash
        ((WORD *)&_max_valid_hash)[threadIdx.x] =
            ((WORD *)max_valid_hash)[threadIdx.x];
    }
    __syncthreads();
#else                               // put it into thread-local registers
    uint256_t _max_valid_hash;
    // populate _max_valid_hash
#pragma unroll 32 / sizeof(WORD)
    for (int i = 0; i < SHA256_BLOCK_SIZE / sizeof(WORD); i++) {
        ((WORD *) &_max_valid_hash)[i] = ((WORD *) max_valid_hash)[i];
    }
#endif

    if (thread >= n_batch_device) {
        return;
    }

    uint128_t nonce = *((uint128_t *) init_nonce) + thread;
    BYTE in[sizeof(uint128_t) +
            SHA256_BLOCK_SIZE];  // nonce + const_in = 384 bytes
#pragma unroll 32 / sizeof(WORD)
    for (int i = 0; i < SHA256_BLOCK_SIZE / sizeof(WORD); i++) {
        ((WORD *) in)[i + sizeof(uint128_t) / sizeof(WORD)] = ((WORD *) const_in)[i];
    }

    uint256_t out;
    CUDA_SHA256_CTX ctx;
    uint128_t *_nonce_out = (uint128_t *) nonce_out;
    bool success = false;
    while (1) {
        // write nonce to first bytes of in
        *((uint128_t *) in) = nonce;
        // sha256 with little endian output
        cuda_sha256_init(&ctx);
        cuda_sha256_update(&ctx, in, sizeof(in));
        cuda_sha256_final(&ctx, (BYTE *) &out);
        // check if nonce is valid
        if (cuda_u256_lte(out, _max_valid_hash)) {
            success = atomicCAS((WORD *) nonce_out, 0, 1) == 0;
            break;
        }
        if (*_nonce_out != 0) break;
        for (int i = 0; i < sizeof(uint128_t) / sizeof(WORD); i++) {
            ((WORD *) &nonce)[i] = hiprand(&state);
        }
    }

    // without syncthreads, if another thread was just doing the atomicCAS and I
    // am currently writing into nonce_out, there is a tiny chance there could be
    // interference. let's not risk this.
    __syncthreads();
    if (success) *((uint128_t *)nonce_out) = nonce;
}
